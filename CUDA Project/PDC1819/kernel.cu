#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>


// To size tou pinaka
#define size 5

__global__ void average(int *a, int *b, int *c) {

	// Initialize 2 topikes metavlites gia na vro to sinolo kai meta to mean
	int totalA = 0, totalB = 0;

	// Loop gia na paroume to sinolo tou A,B
	for (int i = 0; i < size; i++) {
		totalA +=  a[threadIdx.x*size + i];		//Ypologizoume to sinolo tis ka8e stilis
		totalB +=  b[threadIdx.x*size + i];
	}
	c[threadIdx.x] = totalA / 5 + totalB / 5; //Epita ta diairoume /5 kai ta pername sto C
}

int main(void) {
	int A[size][size];      //Dimiourgo tous pinakes A,B,C
	int B[size][size];
	int C[size];
	int *dev_a;      //Dimiourgoume device copies tou a,b,c (pointers)
	int *dev_b;		//gia na stiloume ta dedomena stin GPU
	int *dev_c;

	// Gemizo tous Pinakes A & B
	for (int i = 0; i < size; i++) {
		printf("C[%i]=\n", i);
		for (int j = 0; j < size; j++) {
			A[i][j] = rand() % 10;
			B[i][j] = rand() % 10;
			printf("A:%i, ", A[i][j]);
			printf("B:%i,\n", B[i][j]);
		}
		printf("\n");
	}

	// Dilonoume to megethos tou pinaka pou 8a xriastoume
	int size_2d = size * size * sizeof(int);
	int size_c = size * sizeof(int);

	// Desmeuo mnimi sto sistima
	hipMalloc(&dev_a, size_2d);
	hipMalloc(&dev_b, size_2d);
	hipMalloc(&dev_c, size_c);

	// Copy ton dedomenon stin mnimi tis GPU (meso pointers)
	hipMemcpy(dev_a, A, size_2d, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, B, size_2d, hipMemcpyHostToDevice);
	hipMemcpy(dev_c, C, size_c, hipMemcpyHostToDevice);

	// Kalo tin kerner
	average <<< size, size >>> (dev_a, dev_b, dev_c);

	// Travao to output piso stin CPU
	hipMemcpy(C, dev_c, size_c, hipMemcpyDeviceToHost);

	// Ta emfanizo
	printf("\n");
	for (int i = 0; i < size; i++) {
		printf("C[%i]= %i\n", i, C[i]);
	}

	// Eleutherono tin mnimi
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
}